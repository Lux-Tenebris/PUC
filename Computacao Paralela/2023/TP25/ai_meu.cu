
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void scan_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  double x;
  __shared__ double p[1024];
  if(b+t < width)
    p[t] = a[b+t];
  __syncthreads();

  for (int i = 1; i < blockDim.x; i *= 2) {
    if(t >= i)
      x = p[t] + p[t-i];
    __syncthreads();
    if(t >= i)
      p[t] = x;
    __syncthreads();
  }
  if(b + t < width)
    a[b+t] = p[t];
  if(t == blockDim.x-1)
    s[blockIdx.x+1] = a[b+t];
}

__global__ void add_cuda(double *a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  if(b+t < width)
    a[b+t] += s[blockIdx.x];
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_s, s_size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  dim3 dimBlock(block_size, 1, 1);
  dim3 dimGrid(num_blocks, 1, 1);

  scan_cuda<<<dimGrid, dimBlock>>>(d_a, d_s, width);

  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];

  hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);

  add_cuda<<<dimGrid, dimBlock>>>(d_a, d_s, width);

  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
